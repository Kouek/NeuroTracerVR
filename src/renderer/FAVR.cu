#include "hip/hip_runtime.h"
#include "vr_renderer.h"

#include <VolumeSlicer/helper.hpp>

#include <>

using namespace kouek;

hipArray_t d_subsampleSurfArr2[2];
hipSurfaceObject_t d_subsampleSurf2[2];
__constant__ hipSurfaceObject_t dc_subsampleSurf2[2];

hipArray_t d_subsampleLookupTexArrs[VRRenderer::MAX_SUBSAMPLE_LEVEL];
hipTextureObject_t d_subsampleLookupTexes[VRRenderer::MAX_SUBSAMPLE_LEVEL];
__constant__ hipTextureObject_t
    dc_subsampleLookupTexes[VRRenderer::MAX_SUBSAMPLE_LEVEL];

hipArray_t d_reconsLookupTexArrs[VRRenderer::MAX_SUBSAMPLE_LEVEL];
hipTextureObject_t d_reconsLookupTexes[VRRenderer::MAX_SUBSAMPLE_LEVEL];
__constant__ hipTextureObject_t
    dc_reconsLookupTexes[VRRenderer::MAX_SUBSAMPLE_LEVEL];

__global__ void createSubsampleTexKernel(uint2 *d_output, uint8_t FAVRLvl,
                                         uint32_t w, uint32_t h) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    if (renderXY.x >= w || renderXY.y >= h)
        return;
    size_t flatIdx = (size_t)renderXY.y * w + renderXY.x;

    uint32_t startY = 0, endY = w;
    float hfW = .5f * w;
    float hfWSqr = hfW * hfW;
    float fuleyeIdxesHfSz = .5f * h;
    glm::vec2 cntr{w * .5f};
    glm::vec2 pos{renderXY.x + .5f, renderXY.y + .5f};
    for (uint8_t stage = 0; stage < FAVRLvl; ++stage) {
        if (renderXY.y >= endY) {
            startY = endY;
            endY += w;
            cntr.y += w;
            continue;
        }

        auto scale = 1.f - (float)stage / FAVRLvl;
        auto invScale = 1.f / scale;
        auto scaleSqr = (1.f - scale) * (1.f - scale);

        auto d = pos - cntr;
        auto sqr = glm::dot(d, d);
        auto lowerSqr = stage == 0 ? 0 : hfWSqr * scaleSqr;
        if (sqr < lowerSqr || sqr > (stage == FAVRLvl - 1 ? INFINITY : hfWSqr))
            d_output[flatIdx] = {VRRenderer::SUBSAMPLE_MAP_TO_EMPTY,
                                 VRRenderer::SUBSAMPLE_MAP_TO_EMPTY};
        else {
            auto d0 = glm::normalize(d);
            d = invScale * (d - hfW * (1.f - scale) * d0);
            d0 = hfW * stage * d0;
            glm::vec2 targetPx = {floorf(fuleyeIdxesHfSz + d0.x + d.x),
                                  floorf(fuleyeIdxesHfSz + d0.y + d.y)};
            targetPx = glm::clamp(targetPx, glm::vec2{0}, glm::vec2{h});
            d_output[flatIdx] = {(uint32_t)targetPx.x, (uint32_t)targetPx.y};
        }
        break;
    }
}

__global__ void createReconsTexKernel(uint2 *d_output, uint8_t FAVRLvl,
                                      uint32_t sz, uint32_t subsampleW) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    if (renderXY.x >= sz || renderXY.y >= sz)
        return;
    size_t flatIdx = (size_t)renderXY.y * sz + renderXY.x;

    glm::vec2 d{renderXY.x + .5f - sz * .5f, renderXY.y + .5f - sz * .5f};
    auto sqr = glm::dot(d, d);
    auto hfSubsampleW = subsampleW * .5f;
    glm::vec2 subsampleCntr{hfSubsampleW};
    auto rad = hfSubsampleW;
    auto radSqr = rad * rad;
    for (uint8_t stage = 0; stage < FAVRLvl; ++stage) {
        auto scale = 1.f - (float)stage / FAVRLvl;
        auto scaleSqr = scale * scale;
        if (stage != FAVRLvl - 1 && sqr >= radSqr) {
            subsampleCntr.y += subsampleW;
            rad += hfSubsampleW;
            radSqr = rad * rad;
            continue;
        }

        auto d0 = glm::normalize(d);
        d = scale * (d - hfSubsampleW * stage * d0);
        d0 = hfSubsampleW * (1.f - scale) * d0;
        d_output[flatIdx] = {(uint32_t)floorf(subsampleCntr.x + d0.x + d.x),
                             (uint32_t)floorf(subsampleCntr.y + d0.y + d.y)};
        break;
    }
}

static const uint8_t TEST_TIMES = 20;
__global__ void dismissGapKernel(uint2 *d_output, uint8_t FAVRLvl, uint32_t sz,
                                 uint32_t subsampleW) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    if (renderXY.x >= sz || renderXY.y >= sz)
        return;
    size_t flatIdx = (size_t)renderXY.y * sz + renderXY.x;

    auto FAVRIdx = FAVRLvl - 1;
    {
        auto subsampleTexPos = d_output[flatIdx];
        auto subsamplePos = tex2D<uint2>(dc_subsampleLookupTexes[FAVRIdx],
                                         subsampleTexPos.x, subsampleTexPos.y);
        if (subsamplePos.x != VRRenderer::SUBSAMPLE_MAP_TO_EMPTY)
            return;
    }

    glm::vec2 d{renderXY.x + .5f - sz * .5f, renderXY.y + .5f - sz * .5f};
    d = glm::normalize(d);
    if (fabsf(d.x) > fabsf(d.y))
        d *= 1.f / fabsf(d.x);
    else
        d *= 1.f / fabsf(d.y);

    auto sqr = glm::dot(d, d);
    auto hfSubsampleW = subsampleW * .5f;
    auto rad = hfSubsampleW;
    auto radSqr = rad * rad;
    for (uint8_t stage = 0; stage < FAVRLvl; ++stage) {
        if (stage != FAVRLvl - 1 && sqr >= radSqr) {
            rad += hfSubsampleW;
            radSqr = rad * rad;
            continue;
        }
        glm::vec2 newRndrXY = {(float)renderXY.x + .5f - d.x,
                               (float)renderXY.y + .5f - d.y};
        bool found = false;
        for (uint8_t inStep = 0; inStep < TEST_TIMES; ++inStep) {
            size_t newFlatIdx =
                (size_t)floorf(newRndrXY.y) * sz + (size_t)floorf(newRndrXY.x);
            auto newSubsampleTexPos = d_output[newFlatIdx];
            auto subsamplePos =
                tex2D<uint2>(dc_subsampleLookupTexes[FAVRIdx],
                             newSubsampleTexPos.x, newSubsampleTexPos.y);
            if (subsamplePos.x != VRRenderer::SUBSAMPLE_MAP_TO_EMPTY) {
                d_output[flatIdx] = newSubsampleTexPos;
                found = true;
                break;
            }
            newRndrXY -= d;
        }
        if (found)
            break;
        newRndrXY = {(float)renderXY.x + .5f + d.x,
                     (float)renderXY.y + .5f + d.y};
        for (uint8_t ouStep = 0; ouStep < TEST_TIMES; ++ouStep) {
            size_t newFlatIdx =
                (size_t)floorf(newRndrXY.y) * sz + (size_t)floorf(newRndrXY.x);
            auto newSubsampleTexPos = d_output[newFlatIdx];
            auto subsamplePos =
                tex2D<uint2>(dc_subsampleLookupTexes[FAVRIdx],
                             newSubsampleTexPos.x, newSubsampleTexPos.y);
            if (subsamplePos.x != VRRenderer::SUBSAMPLE_MAP_TO_EMPTY) {
                d_output[flatIdx] = newSubsampleTexPos;
                break;
            }
            newRndrXY += d;
        }
        break;
    }
}

void VRRenderer::createSubsampleAndReconsTex() {
    assert(renderSz.x == renderSz.y);
    float hfSz = renderSz.y * .5f;

    hipResourceDesc rsrcDesc;
    memset(&rsrcDesc, 0, sizeof(rsrcDesc));
    rsrcDesc.resType = hipResourceTypeArray;

    for (uint8_t eyeIdx = 0; eyeIdx < 2; ++eyeIdx) {
        if (d_subsampleSurfArr2[eyeIdx] != nullptr) {
            CUDA_RUNTIME_API_CALL(
                hipDestroySurfaceObject(d_subsampleSurf2[eyeIdx]));
            CUDA_RUNTIME_API_CALL(hipFreeArray(d_subsampleSurfArr2[eyeIdx]));
        }
        hipChannelFormatDesc chnnlDesc = hipCreateChannelDesc<uchar4>();
        CUDA_RUNTIME_API_CALL(hipMallocArray(
            &d_subsampleSurfArr2[eyeIdx], &chnnlDesc, renderSz.x, renderSz.y));

        rsrcDesc.res.array.array = d_subsampleSurfArr2[eyeIdx];
        CUDA_RUNTIME_API_CALL(
            hipCreateSurfaceObject(&d_subsampleSurf2[eyeIdx], &rsrcDesc));
    }
    CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dc_subsampleSurf2), d_subsampleSurf2, sizeof(d_subsampleSurf2[0]) * 2));

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.normalizedCoords = false;
    texDesc.readMode = hipReadModeElementType;

    dim3 threadPerBlock{16, 16};
    for (uint8_t lvl = 1; lvl <= MAX_SUBSAMPLE_LEVEL; ++lvl) {
        auto idx = lvl - 1;
        auto subsampleHfW = hfSz / lvl;
        uint32_t w = FAVRSubsampleWidths[idx] = subsampleHfW * 2.f;

        if (d_subsampleLookupTexArrs[idx] != nullptr)
            CUDA_RUNTIME_API_CALL(hipFreeArray(d_subsampleLookupTexArrs[idx]));
        hipChannelFormatDesc chnnlDesc = hipCreateChannelDesc<uint2>();
        CUDA_RUNTIME_API_CALL(hipMallocArray(&d_subsampleLookupTexArrs[idx],
                                              &chnnlDesc, w, renderSz.y));

        rsrcDesc.res.array.array = d_subsampleLookupTexArrs[idx];
        if (d_subsampleLookupTexes[idx] != 0)
            CUDA_RUNTIME_API_CALL(
                hipDestroyTextureObject(d_subsampleLookupTexes[idx]));
        CUDA_RUNTIME_API_CALL(hipCreateTextureObject(
            &d_subsampleLookupTexes[idx], &rsrcDesc, &texDesc, nullptr));

        uint2 *d_tmp;
        auto cpySz = sizeof(uint2) * w * renderSz.y;
        CUDA_RUNTIME_API_CALL(hipMalloc(&d_tmp, cpySz));

        dim3 blockPerGrid{
            (w + threadPerBlock.x - 1) / threadPerBlock.x,
            (renderSz.y + threadPerBlock.y - 1) / threadPerBlock.y, 2};
        createSubsampleTexKernel<<<blockPerGrid, threadPerBlock>>>(
            d_tmp, lvl, w, renderSz.y);

        CUDA_RUNTIME_API_CALL(hipMemcpyToArray(d_subsampleLookupTexArrs[idx],
                                                0, 0, d_tmp, cpySz,
                                                hipMemcpyDeviceToDevice));
        CUDA_RUNTIME_API_CALL(hipFree(d_tmp));
    }
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_subsampleLookupTexes), d_subsampleLookupTexes,
                           sizeof(hipTextureObject_t) * MAX_SUBSAMPLE_LEVEL));

    uint2 *d_tmp;
    auto cpySz = sizeof(uint2) * renderSz.x * renderSz.y;
    CUDA_RUNTIME_API_CALL(hipMalloc(&d_tmp, cpySz));
    for (uint8_t lvl = 1; lvl <= MAX_SUBSAMPLE_LEVEL; ++lvl) {
        auto idx = lvl - 1;

        if (d_reconsLookupTexArrs[idx] != nullptr)
            CUDA_RUNTIME_API_CALL(hipFreeArray(d_reconsLookupTexArrs[idx]));
        hipChannelFormatDesc chnnlDesc = hipCreateChannelDesc<uint2>();
        CUDA_RUNTIME_API_CALL(hipMallocArray(
            &d_reconsLookupTexArrs[idx], &chnnlDesc, renderSz.x, renderSz.y));

        rsrcDesc.res.array.array = d_reconsLookupTexArrs[idx];
        if (d_reconsLookupTexes[idx] != 0)
            CUDA_RUNTIME_API_CALL(
                hipDestroyTextureObject(d_reconsLookupTexes[idx]));
        CUDA_RUNTIME_API_CALL(hipCreateTextureObject(
            &d_reconsLookupTexes[idx], &rsrcDesc, &texDesc, nullptr));

        dim3 blockPerGrid{
            (renderSz.x + threadPerBlock.x - 1) / threadPerBlock.x,
            (renderSz.y + threadPerBlock.y - 1) / threadPerBlock.y, 2};
        createReconsTexKernel<<<blockPerGrid, threadPerBlock>>>(
            d_tmp, lvl, renderSz.x, FAVRSubsampleWidths[idx]);
        dismissGapKernel<<<blockPerGrid, threadPerBlock>>>(
            d_tmp, lvl, renderSz.x, FAVRSubsampleWidths[idx]);
        CUDA_RUNTIME_API_CALL(hipMemcpyToArray(d_reconsLookupTexArrs[idx], 0,
                                                0, d_tmp, cpySz,
                                                hipMemcpyDeviceToDevice));
    }
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_reconsLookupTexes), d_reconsLookupTexes,
                           sizeof(hipTextureObject_t) * MAX_SUBSAMPLE_LEVEL));

    CUDA_RUNTIME_API_CALL(hipFree(d_tmp));
}
