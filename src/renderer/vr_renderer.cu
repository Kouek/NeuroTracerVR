#include "hip/hip_runtime.h"
#include "vr_renderer.h"

#include <VolumeSlicer/helper.hpp>

#include <cuda_gl_interop.h>
#include <>

using namespace kouek;

static hipArray_t d_preIntTFArray;
static hipTextureObject_t d_preIntTF;
__constant__ hipTextureObject_t dc_preIntTF;
__constant__ VRRenderer::LightParam dc_lightParam;
__constant__ VRRenderer::RenderParam dc_renderParam;
__constant__ VRRenderer::CameraParam dc_cameraParam;
__constant__ VRRenderer::ProjectionParam dc_projectionParam;

hipGraphicsResource_t outputSurfRsrc2[2];
hipGraphicsResource_t inputDepTexRsrc2[2];

void VRRenderer::SetCUDAxGLParam(const CUDAxGLParam &param) {
    size_t pxNum = param.renderSz.x * param.renderSz.y;

    for (uint8_t eyeIdx = 0; eyeIdx < 2; ++eyeIdx) {
        CUDA_RUNTIME_API_CALL(hipGraphicsGLRegisterImage(
            &outputSurfRsrc2[eyeIdx], param.outputTex2[eyeIdx], GL_TEXTURE_2D,
            hipGraphicsRegisterFlagsWriteDiscard));

        CUDA_RUNTIME_API_CALL(hipGraphicsGLRegisterImage(
            &inputDepTexRsrc2[eyeIdx], param.inputDepTex2[eyeIdx],
            GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
    }
}

void VRRenderer::SetLightParam(const LightParam &param) {
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_lightParam), &param, sizeof(LightParam)));
}

void VRRenderer::SetRenderParam(const RenderParam &param) {
    assert(param.FAVRLvl <= MAX_SUBSAMPLE_LEVEL);
    FAVRLvl = param.FAVRLvl;
    FAVRIdx = FAVRLvl - 1;
    if (renderSz != param.renderSz) {
        renderSz = param.renderSz;
        createSubsampleAndReconsTex();
    }
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_renderParam), &param, sizeof(RenderParam)));
}

void VRRenderer::SetTransferFunction(const std::array<glm::vec4, 256> &tfPnts) {
    std::vector<glm::vec4> preIntTF(256 * 256);
    float rayStep = 1.f;
    for (int32_t sb = 0; sb < 256; ++sb)
        for (int32_t sf = 0; sf <= sb; sf++) {
            int32_t offset = sf != sb;
            int32_t n = 20 + std::abs(sb - sf);
            float stepWidth = rayStep / n;
            glm::vec4 rgba{0};
            for (int32_t i = 0; i < n; i++) {
                float s = sf + (sb - sf) * (float)i / n;
                float sFrac = s - std::floor(s);
                float opacity = (tfPnts[(int32_t)s].a * (1.f - sFrac) +
                                 tfPnts[(int32_t)s + offset].a * sFrac) *
                                stepWidth;
                float temp = std::exp(-rgba.a) * opacity;
                rgba.r += (tfPnts[(int32_t)s].r * (1.f - sFrac) +
                           tfPnts[(int32_t)s + offset].r * sFrac) *
                          temp;
                rgba.g += (tfPnts[(int32_t)s].g * (1.f - sFrac) +
                           tfPnts[(int32_t)s + offset].g * sFrac) *
                          temp;
                rgba.b += (tfPnts[(int32_t)s].b * (1.f - sFrac) +
                           tfPnts[(int32_t)s + offset].b * sFrac) *
                          temp;
                rgba.a += opacity;
            }
            preIntTF[sf * 256 + sb].r = preIntTF[sb * 256 + sf].r = rgba.r;
            preIntTF[sf * 256 + sb].g = preIntTF[sb * 256 + sf].g = rgba.g;
            preIntTF[sf * 256 + sb].b = preIntTF[sb * 256 + sf].b = rgba.b;
            preIntTF[sf * 256 + sb].a = preIntTF[sb * 256 + sf].a =
                1.f - std::exp(-rgba.a);
        }

    if (d_preIntTFArray == nullptr) {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
        CUDA_RUNTIME_API_CALL(
            hipMallocArray(&d_preIntTFArray, &channelDesc, 256, 256));
        hipResourceDesc texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array = d_preIntTFArray;
        hipTextureDesc texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = true;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeClamp;
        texDescr.addressMode[1] = hipAddressModeClamp;
        texDescr.readMode = hipReadModeElementType;
        CUDA_RUNTIME_API_CALL(
            hipCreateTextureObject(&d_preIntTF, &texRes, &texDescr, NULL));
        CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dc_preIntTF), &d_preIntTF,
                                                 sizeof(hipTextureObject_t)));
    }
    CUDA_RUNTIME_API_CALL(hipMemcpy2DToArray(
        d_preIntTFArray, 0, 0, preIntTF.data(), sizeof(glm::vec4) * 256,
        sizeof(glm::vec4) * 256, 256, hipMemcpyHostToDevice));
}

void VRRenderer::SetCameraParam(const CameraParam &param) {
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_cameraParam), &param, sizeof(CameraParam)));
}

void VRRenderer::SetProjectionParam(const ProjectionParam &param) {
    CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dc_projectionParam), &param,
                                             sizeof(ProjectionParam)));
}
