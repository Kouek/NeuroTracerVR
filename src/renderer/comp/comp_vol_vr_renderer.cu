#include "hip/hip_runtime.h"
#include "comp_vol_vr_renderer.h"

#include <cassert>
#include <chrono>

#include <VolumeSlicer/helper.hpp>

#include <>

using namespace kouek;

extern __constant__ hipTextureObject_t dc_preIntTF;
extern __constant__ VRRenderer::LightParam dc_lightParam;
extern __constant__ VRRenderer::RenderParam dc_renderParam;
extern __constant__ VRRenderer::CameraParam dc_cameraParam;
extern __constant__ VRRenderer::ProjectionParam dc_projectionParam;

extern hipGraphicsResource_t outputSurfRsrc2[2];
extern hipGraphicsResource_t inputDepTexRsrc2[2];

extern __constant__ hipSurfaceObject_t dc_subsampleSurf2[2];
extern __constant__ hipTextureObject_t
    dc_subsampleLookupTexes[VRRenderer::MAX_SUBSAMPLE_LEVEL];
extern __constant__ hipTextureObject_t
    dc_reconsLookupTexes[VRRenderer::MAX_SUBSAMPLE_LEVEL];

__constant__ CompVolVRRenderer::RendererParam dc_rendererParam;
__constant__ CompVolVRRenderer::VolumeParam dc_volumeParam;

__constant__ uint32_t dc_blockOffsets[CompVolVRRenderer::MAX_LOD + 1];
__constant__ hipTextureObject_t
    dc_textures[CompVolVRRenderer::MAX_TEX_UNIT_NUM];

static const dim3 threadPerBlock{16, 16};

static uint32_t *d_mappingTable = nullptr;
__constant__ size_t dc_mappingTableSize = 0;
__constant__ glm::uvec4 *dc_mappingTableStride4 = nullptr;

static hipTextureDesc inputDepTexRsrcDesc;

static hipStream_t renderStream = nullptr;

void uploadBlockOffset(const uint32_t *dat, size_t num) {
    assert(num <= CompVolVRRenderer::MAX_LOD + 1);
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_blockOffsets), dat, sizeof(uint32_t) * num));
}

void uploadTextureObject(const hipTextureObject_t *tex, size_t num) {
    assert(num <= CompVolVRRenderer::MAX_TEX_UNIT_NUM);
    CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dc_textures), tex, sizeof(hipTextureObject_t) * num));
}

void uploadMappingTable(const uint32_t *dat, size_t size) {
    if (d_mappingTable == nullptr) {
        hipMalloc(&d_mappingTable, size);
        // cpy uint32_t ptr to uint4 ptr
        CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
            dc_mappingTableStride4), &d_mappingTable, sizeof(glm::uvec4 *)));
    }
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_mappingTableSize), &size, sizeof(size_t)));
    CUDA_RUNTIME_API_CALL(
        hipMemcpy(d_mappingTable, dat, size, hipMemcpyHostToDevice));
}

CompVolVRRenderer::CompVolVRRenderer(const RendererParam &param)
    : rendererParam(param) {
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_rendererParam), &rendererParam,
                           sizeof(CompVolVRRenderer::RendererParam)));

    memset(&inputDepTexRsrcDesc, 0, sizeof(inputDepTexRsrcDesc));
    inputDepTexRsrcDesc.normalizedCoords = 0;
    inputDepTexRsrcDesc.filterMode = hipFilterModePoint;
    inputDepTexRsrcDesc.addressMode[0] = hipAddressModeClamp;
    inputDepTexRsrcDesc.addressMode[1] = hipAddressModeClamp;
    inputDepTexRsrcDesc.readMode = hipReadModeElementType;
}

void CompVolVRRenderer::SetCameraParam(const CameraParam &param) {
    VRRenderer::SetCameraParam(param);

    static auto start = std::chrono::system_clock::now();
    auto now = std::chrono::system_clock::now();
    if (std::chrono::duration_cast<std::chrono::milliseconds>(now - start) >=
        static_cast<std::chrono::milliseconds>(2500)) {
        start = now;
    } else
        return; // load block every x ms

    loadBlocks.clear();
    unloadBlocks.clear();
    {
        vs::Pyramid pyramid(
            param.headPos,
            param.headPos + param.rotation * camPyramidParam.pos4[2],
            param.headPos + param.rotation * camPyramidParam.pos4[3],
            param.headPos + param.rotation * camPyramidParam.pos4[0],
            param.headPos + param.rotation * camPyramidParam.pos4[1]);
        auto obb = pyramid.getOBB();
        auto aabb = obb.getAABB();

        // AABB filter first
        for (auto &blockAABB : blockAABBs)
            if (aabb.intersect(blockAABB.second))
                currNeedBlocks.emplace(
                    std::array{blockAABB.first[0], blockAABB.first[1],
                               blockAABB.first[2], (uint32_t)0});
        // OBB filter then
        for (auto itr = currNeedBlocks.begin(); itr != currNeedBlocks.end();)
            if (!obb.intersect_obb(
                    blockAABBs[std::array{(*itr)[0], (*itr)[1], (*itr)[2]}]
                        .convertToOBB()))
                itr = currNeedBlocks.erase(itr);
            else
                ++itr;
    }

    // loadBlocks = currNeedBlocks - (old)needBlocks
    for (auto &e : currNeedBlocks)
        if (needBlocks.find(e) == needBlocks.end())
            loadBlocks.insert(e);

    // unloadBlocks = (old)needBlocks - currNeedBlocks
    for (auto &e : needBlocks)
        if (currNeedBlocks.find(e) == currNeedBlocks.end())
            unloadBlocks.insert(e);

    needBlocks = std::move(currNeedBlocks);
    if (loadBlocks.size() > 0 || unloadBlocks.size() > 0) {
        // loadBlocks = loadBlocks - cachedBlocks
        decltype(loadBlocks) tmp;
        for (auto &e : loadBlocks) {
            bool cached = blockCache->SetCachedBlockValid(e);
            if (!cached)
                tmp.insert(e);
        }
        loadBlocks = std::move(tmp);

        for (auto &e : unloadBlocks)
            blockCache->SetBlockInvalid(e);

        volume.get()->PauseLoadBlock();

        if (!needBlocks.empty()) {
            std::vector<std::array<uint32_t, 4>> targets;
            targets.reserve(needBlocks.size());
            for (auto &e : needBlocks)
                targets.push_back(e);
            volume.get()->ClearBlockInQueue(targets);
        }
        for (auto &e : loadBlocks)
            volume.get()->SetRequestBlock(e);
        for (auto &e : unloadBlocks)
            volume.get()->EraseBlockInRequest(e);

        volume.get()->StartLoadBlock();
    }
}

void CompVolVRRenderer::Render(RenderTarget renderTarget) {
    for (auto &e : needBlocks) {
        auto volumeBlock = volume.get()->GetBlock(e);
        if (volumeBlock.valid) {
            blockCache->UploadVolumeBlock(e,
                                          volumeBlock.block_data->GetDataPtr(),
                                          volumeBlock.block_data->GetSize());
            volumeBlock.Release();
        }
    }
    auto &mappingTable = blockCache->GetMappingTable();
    uploadMappingTable(mappingTable.data(),
                       sizeof(uint32_t) * mappingTable.size());

    if (renderStream == nullptr)
        CUDA_RUNTIME_API_CALL(hipStreamCreate(&renderStream));
    render(renderTarget);
}

void CompVolVRRenderer::SetVolume(std::shared_ptr<vs::CompVolume> volume,
                                  const glm::vec3 &spaces) {
    this->volume = volume;
    VolumeParam volumeParam;
    {
        auto blockDimInfo = volume->GetBlockDim();
        auto blockLenInfo = volume->GetBlockLength();
        volumeParam.blockLength = blockLenInfo[0];
        volumeParam.LOD0BlockDim = {blockDimInfo[0][0], blockDimInfo[0][1],
                                    blockDimInfo[0][2]};
        volumeParam.noPaddingBlockLength =
            blockLenInfo[0] - 2 * blockLenInfo[1];
        volumeParam.padding = blockLenInfo[1];
        volumeParam.spaces = spaces;

        CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dc_volumeParam), &volumeParam,
                                                 sizeof(VolumeParam)));
    }

    blockCache = vs::CUDAVolumeBlockCache::Create();
    blockCache->SetCacheBlockLength(volume->GetBlockLength()[0]);
    blockCache->SetCacheCapacity(
        rendererParam.texUnitNum, rendererParam.texUnitShape.x,
        rendererParam.texUnitShape.y, rendererParam.texUnitShape.z);
    blockCache->CreateMappingTable(volume->GetBlockDim());
    {
        // map lod to flat({ lod,0,0,0 }),
        // which is the first Voxel idx of LOD lod
        auto &lodMappingTableOffsets = blockCache->GetLodMappingTableOffset();
        uint32_t maxLOD = 0, minLOD = std::numeric_limits<uint32_t>::max();
        for (auto &e : lodMappingTableOffsets) {
            if (e.first < minLOD)
                minLOD = e.first;
            if (e.first > maxLOD)
                maxLOD = e.first;
        }
        maxLOD--; // in lodMappingTableOffsets, Key ranges [0, MAX_LOD + 1]

        // map lod(idx of vector) to flat({ lod,0,0,0 }) / 4,
        // which is the first Block idx of LOD lod
        std::vector<uint32_t> blockOffsets((size_t)maxLOD + 1, 0);
        for (auto &e : lodMappingTableOffsets)
            // in lodMappingTableOffsets, Key ranges [0, MAX_LOD + 1],
            // while in blockOffsets, Key ranges [0, MAX_LOD]
            if (e.first <= maxLOD)
                blockOffsets[e.first] = e.second / 4;

        // upload to CUDA
        uploadBlockOffset(blockOffsets.data(), blockOffsets.size());
    }
    {
        auto &texObj = blockCache->GetCUDATextureObjects();
        uploadTextureObject(texObj.data(), texObj.size());
    }

    blockAABBs.clear(); // avoid conflict caused by reset
    for (uint32_t z = 0; z < volumeParam.LOD0BlockDim.z; ++z)
        for (uint32_t y = 0; y < volumeParam.LOD0BlockDim.y; ++y)
            for (uint32_t x = 0; x < volumeParam.LOD0BlockDim.x; ++x)
                blockAABBs.emplace(
                    std::piecewise_construct,
                    std::forward_as_tuple(std::array{x, y, z}),
                    std::forward_as_tuple(
                        glm::vec3{
                            x * volumeParam.noPaddingBlockLength * spaces.x,
                            y * volumeParam.noPaddingBlockLength * spaces.y,
                            z * volumeParam.noPaddingBlockLength * spaces.z},
                        glm::vec3{(x + 1) * volumeParam.noPaddingBlockLength *
                                      spaces.x,
                                  (y + 1) * volumeParam.noPaddingBlockLength *
                                      spaces.y,
                                  (z + 1) * volumeParam.noPaddingBlockLength *
                                      spaces.z},
                        // dummy in this program
                        std::array<uint32_t, 4>()));
}

void CompVolVRRenderer::SetCameraPyramidParam(const CamPyramidParam &param) {
    camPyramidParam = param;
}

__device__ float virtualSampleLOD0(const glm::vec3 &samplePos) {
    // sample pos in Voxel Space -> virtual sample Block idx
    glm::uvec3 vsBlockIdx =
        samplePos / (float)dc_volumeParam.noPaddingBlockLength;

    // virtual sample Block idx -> real sample Block idx (in GPU Mem)
    glm::uvec4 GPUMemBlockIdx;
    {
        size_t flatVSBlockIdx = dc_blockOffsets[0] +
                                vsBlockIdx.z * dc_volumeParam.LOD0BlockDim.y *
                                    dc_volumeParam.LOD0BlockDim.x +
                                vsBlockIdx.y * dc_volumeParam.LOD0BlockDim.x +
                                vsBlockIdx.x;
        if (flatVSBlockIdx >= dc_mappingTableSize)
            return 0;
        GPUMemBlockIdx = dc_mappingTableStride4[flatVSBlockIdx];
    }
    if (((GPUMemBlockIdx.w >> 16) & (0x0000ffff)) != 1)
        // not a valid GPU Mem block
        return 0;

    // sample pos in Voxel Space -> real sample pos (in GPU Mem)
    glm::vec3 GPUMemSamplePos;
    {
        glm::vec3 offsetInNoPaddingBlock =
            samplePos -
            glm::vec3{vsBlockIdx * dc_volumeParam.noPaddingBlockLength};
        GPUMemSamplePos =
            glm::vec3{GPUMemBlockIdx.x, GPUMemBlockIdx.y, GPUMemBlockIdx.z} *
                (float)dc_volumeParam.blockLength +
            offsetInNoPaddingBlock + (float)dc_volumeParam.padding;
        // normolized
        GPUMemSamplePos /= dc_rendererParam.texUnitShape;
    }

    return tex3D<float>(dc_textures[GPUMemBlockIdx.w & 0x0000ffff],
                        GPUMemSamplePos.x, GPUMemSamplePos.y,
                        GPUMemSamplePos.z);
}

__device__ glm::vec4 phongShadingLOD0(const glm::vec3 &rayDrc,
                                      const glm::vec3 &samplePos,
                                      const glm::vec4 &diffuseColor) {
    glm::vec3 N;
    {
        float val1, val2;
        val1 = virtualSampleLOD0(samplePos + glm::vec3{1.f, 0, 0});
        val2 = virtualSampleLOD0(samplePos - glm::vec3{1.f, 0, 0});
        N.x = val2 - val1;
        val1 = virtualSampleLOD0(samplePos + glm::vec3{0, 1.f, 0});
        val2 = virtualSampleLOD0(samplePos - glm::vec3{0, 1.f, 0});
        N.y = val2 - val1;
        val1 = virtualSampleLOD0(samplePos + glm::vec3{0, 0, 1.f});
        val2 = virtualSampleLOD0(samplePos - glm::vec3{0, 0, 1.f});
        N.z = val2 - val1;
    }
    N = glm::normalize(N);

    glm::vec3 L = {-rayDrc.x, -rayDrc.y, -rayDrc.z};
    glm::vec3 R = L;
    if (glm::dot(N, L) < 0)
        N = -N;

    glm::vec3 ambient = dc_lightParam.ka * diffuseColor;
    glm::vec3 specular =
        glm::vec3(dc_lightParam.ks * powf(fmaxf(dot(N, .5f * (L + R)), 0),
                                          dc_lightParam.shininess));
    glm::vec3 diffuse = dc_lightParam.kd * fmaxf(dot(N, L), 0.f) * diffuseColor;

    return glm::vec4{ambient + specular + diffuse, diffuseColor.a};
}

__device__ void rayIntersectAABB(float *tEnter, float *tExit,
                                 const glm::vec3 &rayOri,
                                 const glm::vec3 &rayDrc, const glm::vec3 &bot,
                                 const glm::vec3 &top) {
    // For  Ori + Drc * t3Bot.x = <Bot.x, 0, 0>
    // Thus t3Bot.x = Bot.x / Drc.x
    // Thus t3Bot.y = Bot.x / Drc.y
    // If  \
	//  \_\|\ 
	//   \_\|
    //      \.t3Bot.x
    //      |\
	//    __|_\.___|
    //      |  \t3Bot.y
    //    __|___\._|_
    //    t3Top.y\ |
    //      |     \.t3Top.x
    //
    // Then t3Min = t3Bot, t3Max = t3Top
    // And  the max of t3Min is tEnter
    // And  the min of t3Max is tExit

    glm::vec3 invRay = 1.f / rayDrc;
    glm::vec3 t3Bot = invRay * (bot - rayOri);
    glm::vec3 t3Top = invRay * (top - rayOri);
    glm::vec3 t3Min{fminf(t3Bot.x, t3Top.x), fminf(t3Bot.y, t3Top.y),
                    fminf(t3Bot.z, t3Top.z)};
    glm::vec3 t3Max{fmaxf(t3Bot.x, t3Top.x), fmaxf(t3Bot.y, t3Top.y),
                    fmaxf(t3Bot.z, t3Top.z)};
    *tEnter = fmaxf(fmaxf(t3Min.x, t3Min.y), fmaxf(t3Min.x, t3Min.z));
    *tExit = fminf(fminf(t3Max.x, t3Max.y), fminf(t3Max.x, t3Max.z));
}

__device__ uchar4 rgbaFloatToUbyte4(glm::vec4 color) {
    color.r = __saturatef(color.r); // clamp to [0.0, 1.0]
    color.g = __saturatef(color.g);
    color.b = __saturatef(color.b);
    color.a = __saturatef(color.a);
    color.r *= 255.f;
    color.g *= 255.f;
    color.b *= 255.f;
    color.a *= 255.f;
    return make_uchar4(color.r, color.g, color.b, color.a);
}

__global__ void renderKernel(hipTextureObject_t d_inputDepTexLft,
                             hipTextureObject_t d_inputDepTexRht,
                             hipSurfaceObject_t d_outputSurfLft,
                             hipSurfaceObject_t d_outputSurfRht) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    if (renderXY.x >= dc_renderParam.renderSz.x ||
        renderXY.y >= dc_renderParam.renderSz.y)
        return;

    glm::vec3 rayDrc;
    float t, tExit;
    {
        auto v4 = dc_projectionParam.unProjection2[blockIdx.z] *
                  glm::vec4{2.f * renderXY.x / dc_renderParam.renderSz.x - 1.f,
                            2.f * renderXY.y / dc_renderParam.renderSz.y - 1.f,
                            1.f, 1.f};
        rayDrc = v4;
        rayDrc = glm::normalize(rayDrc);
        auto absRayDrcZ = fabsf(rayDrc.z);

        t = dc_projectionParam.nearClip / absRayDrcZ;
        tExit = dc_projectionParam.farClip / absRayDrcZ;

        uchar4 depth4 =
            tex2D<uchar4>(blockIdx.z == 0 ? d_inputDepTexLft : d_inputDepTexRht,
                          renderXY.x, renderXY.y);
        float meshDep =
            dc_projectionParam.projection223[blockDim.z == 0 ? 0 : 1] /
            ((2.f * depth4.x / 255.f - 1.f) +
             dc_projectionParam.projection222[blockDim.z == 0 ? 0 : 1]);
        tExit = glm::min(tExit, meshDep / absRayDrcZ);

        rayDrc = dc_cameraParam.rotation * rayDrc;
    }

    auto rayPos = dc_cameraParam.eyePos2[blockIdx.z] + t * rayDrc;
    auto rayDrcMultStep = dc_renderParam.step * rayDrc;
    auto lastSampleVal = 0.f;
    glm::vec4 color{0};

    decltype(dc_renderParam.maxStepCnt) stepCnt = 0;
    while (true) {
        if (t >= tExit || stepCnt >= dc_renderParam.maxStepCnt)
            break;

        auto samplePos = rayPos / dc_volumeParam.spaces;
        auto currSampleVal = virtualSampleLOD0(samplePos);
        if (currSampleVal > 0) {
            float4 currColor =
                tex2D<float4>(dc_preIntTF, lastSampleVal, currSampleVal);
            lastSampleVal = currSampleVal;
            if (currColor.w > 0) {
                auto shadedColor = phongShadingLOD0(
                    rayDrc, samplePos,
                    {currColor.x, currColor.y, currColor.z, currColor.w});
                color = color + (1.f - color.a) * shadedColor *
                                    glm::vec4{shadedColor.a, shadedColor.a,
                                              shadedColor.a, 1.f};
            }
        }
        if (color.a > .9f)
            break;

        ++stepCnt;
        t += dc_renderParam.step;
        rayPos += rayDrcMultStep;
    }

    constexpr float GAMMA_CORRECT_COEF = 1.f / 2.2f;
    color.r = powf(color.r, GAMMA_CORRECT_COEF);
    color.g = powf(color.g, GAMMA_CORRECT_COEF);
    color.b = powf(color.b, GAMMA_CORRECT_COEF);

    auto outputColor = rgbaFloatToUbyte4(color);

    if (blockIdx.z == 0)
        surf2Dwrite(outputColor, d_outputSurfLft, renderXY.x * 4, renderXY.y);
    else
        surf2Dwrite(outputColor, d_outputSurfRht, renderXY.x * 4, renderXY.y);
}

__global__ void subsampleKernel(hipTextureObject_t d_inputDepTexLft,
                                hipTextureObject_t d_inputDepTexRht,
                                uint32_t subsampleW) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    if (renderXY.x >= subsampleW || renderXY.y >= dc_renderParam.renderSz.y)
        return;

    uint2 subsamplePos;
    {
        auto FAVRIdx = dc_renderParam.FAVRLvl - 1;
        subsamplePos = tex2D<uint2>(dc_subsampleLookupTexes[FAVRIdx],
                                    renderXY.x, renderXY.y);
        if (subsamplePos.x == VRRenderer::SUBSAMPLE_MAP_TO_EMPTY) {
            surf2Dwrite(rgbaFloatToUbyte4(glm::vec4{1.f}),
                        dc_subsampleSurf2[blockIdx.z], renderXY.x * 4,
                        renderXY.y);
            return;
        }
    }

    glm::vec3 rayDrc;
    float t, tExit;
    {
        auto v4 =
            dc_projectionParam.unProjection2[blockIdx.z] *
            glm::vec4{2.f * subsamplePos.x / dc_renderParam.renderSz.x - 1.f,
                      2.f * subsamplePos.y / dc_renderParam.renderSz.y - 1.f,
                      1.f, 1.f};
        rayDrc = v4;
        rayDrc = glm::normalize(rayDrc);

        auto absRayDrcZ = fabsf(rayDrc.z);
        t = dc_projectionParam.nearClip / absRayDrcZ;
        tExit = dc_projectionParam.farClip / absRayDrcZ;

        uchar4 depth4 =
            tex2D<uchar4>(blockIdx.z == 0 ? d_inputDepTexLft : d_inputDepTexRht,
                          subsamplePos.x, subsamplePos.y);
        float meshDep =
            dc_projectionParam.projection223[blockDim.z == 0 ? 0 : 1] /
            ((2.f * depth4.x / 255.f - 1.f) +
             dc_projectionParam.projection222[blockDim.z == 0 ? 0 : 1]);
        tExit = glm::min(tExit, meshDep / absRayDrcZ);

        rayDrc = dc_cameraParam.rotation * rayDrc;
    }

    auto hazeStartSetpCnt =
        dc_renderParam.maxStepCnt *
        CompVolVRRenderer::UNHAZED_RATIO; // step beyond this will be hazed
    auto hazeDltSetpCnt = dc_renderParam.maxStepCnt - hazeStartSetpCnt;
    auto rayPos = dc_cameraParam.eyePos2[blockIdx.z] + t * rayDrc;
    auto rayDrcMultStep = dc_renderParam.step * rayDrc;
    auto lastSampleVal = 0.f;
    glm::vec4 color{0};

    decltype(dc_renderParam.maxStepCnt) stepCnt = 0;
    while (true) {
        if (t >= tExit || stepCnt >= dc_renderParam.maxStepCnt)
            break;

        auto samplePos = rayPos / dc_volumeParam.spaces;
        auto currSampleVal = virtualSampleLOD0(samplePos);
        if (currSampleVal > 0) {
            float4 currColor =
                tex2D<float4>(dc_preIntTF, lastSampleVal, currSampleVal);
            lastSampleVal = currSampleVal;
            if (currColor.w > 0) {
                auto shadedColor = phongShadingLOD0(
                    rayDrc, samplePos,
                    {currColor.x, currColor.y, currColor.z, currColor.w});
                auto hazeK =
                    stepCnt < hazeStartSetpCnt
                        ? 1.f
                        : shadedColor.w *
                              (float)(dc_renderParam.maxStepCnt - stepCnt) /
                              hazeDltSetpCnt;
                color = color + (1.f - color.a) * shadedColor *
                                    glm::vec4{hazeK * shadedColor.a,
                                              hazeK * shadedColor.a,
                                              hazeK * shadedColor.a, 1.f};
            }
        }
        if (color.a > .9f)
            break;

        ++stepCnt;
        t += dc_renderParam.step;
        rayPos += rayDrcMultStep;
    }

    constexpr float GAMMA_CORRECT_COEF = 1.f / 2.2f;
    color.r = powf(color.r, GAMMA_CORRECT_COEF);
    color.g = powf(color.g, GAMMA_CORRECT_COEF);
    color.b = powf(color.b, GAMMA_CORRECT_COEF);

    surf2Dwrite(rgbaFloatToUbyte4(color), dc_subsampleSurf2[blockIdx.z],
                renderXY.x * 4, renderXY.y);
}

__global__ void reconsKernel(hipSurfaceObject_t d_outputSurfLft,
                             hipSurfaceObject_t d_outputSurfRht) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    if (renderXY.x >= dc_renderParam.renderSz.x ||
        renderXY.y >= dc_renderParam.renderSz.y)
        return;

    auto subsampleTexPos =
        tex2D<uint2>(dc_reconsLookupTexes[dc_renderParam.FAVRLvl - 1],
                     renderXY.x, renderXY.y);
    uchar4 subsampleColor;
    surf2Dread(&subsampleColor, dc_subsampleSurf2[blockIdx.z],
               subsampleTexPos.x * 4, subsampleTexPos.y);

    glm::vec4 color;
    color.r = (float)subsampleColor.x;
    color.g = (float)subsampleColor.y;
    color.b = (float)subsampleColor.z;
    color.a = (float)subsampleColor.w;
    for (uint8_t i = 0; i < 4; ++i) {
        glm::uvec2 neighborRenderXY;
        switch (i) {
        case 0:
            neighborRenderXY.x = renderXY.x == 0 ? renderXY.x : renderXY.x - 1;
            break;
        case 1:
            neighborRenderXY.x = renderXY.x == dc_renderParam.renderSz.x - 1
                                     ? renderXY.x
                                     : renderXY.x + 1;
            break;
        default:
            neighborRenderXY.x = renderXY.x;
        }
        switch (i) {
        case 2:
            neighborRenderXY.y = renderXY.y == 0 ? renderXY.y : renderXY.y - 1;
            break;
        case 3:
            neighborRenderXY.y = renderXY.y == dc_renderParam.renderSz.y - 1
                                     ? renderXY.y
                                     : renderXY.y + 1;
            break;
        default:
            neighborRenderXY.y = renderXY.y;
        }
        auto neighborSubsamplePos =
            tex2D<uint2>(dc_reconsLookupTexes[dc_renderParam.FAVRLvl - 1],
                         neighborRenderXY.x, neighborRenderXY.y);
        uchar4 neighborSubsampleColor;
        surf2Dread(&neighborSubsampleColor, dc_subsampleSurf2[blockIdx.z],
                   neighborSubsamplePos.x * 4, neighborSubsamplePos.y);
        color.r += (float)neighborSubsampleColor.x;
        color.g += (float)neighborSubsampleColor.y;
        color.b += (float)neighborSubsampleColor.z;
        color.a += (float)neighborSubsampleColor.w;
    }
    color *= .2f; // avg color
    subsampleColor.x = (uint8_t)color.r;
    subsampleColor.y = (uint8_t)color.g;
    subsampleColor.z = (uint8_t)color.b;
    subsampleColor.w = (uint8_t)color.a;

    if (blockIdx.z == 0)
        surf2Dwrite(subsampleColor, d_outputSurfLft, renderXY.x * 4,
                    renderXY.y);
    else
        surf2Dwrite(subsampleColor, d_outputSurfRht, renderXY.x * 4,
                    renderXY.y);
}

__global__ void testSubsampleKernel(hipSurfaceObject_t d_outputSurfLft,
                                    hipSurfaceObject_t d_outputSurfRht,
                                    uint32_t FAVRWid) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    auto FAVRIdx = dc_renderParam.FAVRLvl - 1;
    if (renderXY.x >= FAVRWid || renderXY.y >= dc_renderParam.renderSz.y)
        return;

    auto subsamplePos =
        tex2D<uint2>(dc_subsampleLookupTexes[FAVRIdx], renderXY.x, renderXY.y);
    auto color =
        subsamplePos.x == VRRenderer::SUBSAMPLE_MAP_TO_EMPTY
            ? glm::vec4{1.f}
            : glm::vec4{(float)subsamplePos.x / dc_renderParam.renderSz.x,
                        (float)subsamplePos.y / dc_renderParam.renderSz.y, 0,
                        1.f};
    if (blockIdx.z == 0)
        surf2Dwrite(rgbaFloatToUbyte4(color), d_outputSurfLft, renderXY.x * 4,
                    renderXY.y);
    else
        surf2Dwrite(rgbaFloatToUbyte4(color), d_outputSurfRht, renderXY.x * 4,
                    renderXY.y);
}

__global__ void testReconsKernel(hipSurfaceObject_t d_outputSurfLft,
                                 hipSurfaceObject_t d_outputSurfRht) {
    glm::uvec2 renderXY{blockIdx.x * blockDim.x + threadIdx.x,
                        blockIdx.y * blockDim.y + threadIdx.y};
    auto FAVRIdx = dc_renderParam.FAVRLvl - 1;
    if (renderXY.x >= dc_renderParam.renderSz.x ||
        renderXY.y >= dc_renderParam.renderSz.y)
        return;

    glm::vec4 color{1.f};
    auto subsampleTexPos =
        tex2D<uint2>(dc_reconsLookupTexes[FAVRIdx], renderXY.x, renderXY.y);
    auto subsamplePos = tex2D<uint2>(dc_subsampleLookupTexes[FAVRIdx],
                                     subsampleTexPos.x, subsampleTexPos.y);
    if (subsamplePos.x != VRRenderer::SUBSAMPLE_MAP_TO_EMPTY) {
        color.r = (float)subsamplePos.x / dc_renderParam.renderSz.x;
        color.g = (float)subsamplePos.y / dc_renderParam.renderSz.y;
        color.b = 0;
    }

    static constexpr auto MOD = 100;
    static constexpr auto HF_WID = 5;
    uint32_t mod[2]{subsamplePos.x % MOD, subsamplePos.y % MOD};
    if (mod[0] < HF_WID || mod[0] > MOD - HF_WID || mod[1] < HF_WID ||
        mod[1] > MOD - HF_WID)
        color = glm::zero<glm::vec4>();

    if (blockIdx.z == 0)
        surf2Dwrite(rgbaFloatToUbyte4(color), d_outputSurfLft, renderXY.x * 4,
                    renderXY.y);
    else
        surf2Dwrite(rgbaFloatToUbyte4(color), d_outputSurfRht, renderXY.x * 4,
                    renderXY.y);
}

void CompVolVRRenderer::render(RenderTarget renderTarget) {
    hipSurfaceObject_t outputSurf2[2];
    hipTextureObject_t inputDepTex2[2];

    hipResourceDesc arrayRsrcDesc;
    memset(&arrayRsrcDesc, 0, sizeof(arrayRsrcDesc));
    arrayRsrcDesc.resType = hipResourceTypeArray;

    for (uint8_t eyeIdx = 0; eyeIdx < 2; ++eyeIdx) {
        hipGraphicsMapResources(1, &outputSurfRsrc2[eyeIdx], renderStream);
        hipGraphicsSubResourceGetMappedArray(&arrayRsrcDesc.res.array.array,
                                              outputSurfRsrc2[eyeIdx], 0, 0);
        hipCreateSurfaceObject(&outputSurf2[eyeIdx], &arrayRsrcDesc);

        hipGraphicsMapResources(1, &inputDepTexRsrc2[eyeIdx], renderStream);
        hipGraphicsSubResourceGetMappedArray(&arrayRsrcDesc.res.array.array,
                                              inputDepTexRsrc2[eyeIdx], 0, 0);
        hipCreateTextureObject(&inputDepTex2[eyeIdx], &arrayRsrcDesc,
                                &inputDepTexRsrcDesc, nullptr);
    }

    switch (renderTarget) {
    case RenderTarget::FullResVol: {
        dim3 blockPerGrid{
            (renderSz.x + threadPerBlock.x - 1) / threadPerBlock.x,
            (renderSz.y + threadPerBlock.y - 1) / threadPerBlock.y, 2};
        renderKernel<<<blockPerGrid, threadPerBlock, 0, renderStream>>>(
            inputDepTex2[0], inputDepTex2[1], outputSurf2[0], outputSurf2[1]);
    } break;
    case RenderTarget::FAVRVol: {
        dim3 threadPerBlock{16, 16};
        dim3 blockPerGrid{
            (FAVRSubsampleWidths[FAVRIdx] + threadPerBlock.x - 1) /
                threadPerBlock.x,
            (renderSz.y + threadPerBlock.y - 1) / threadPerBlock.y, 2};
        subsampleKernel<<<blockPerGrid, threadPerBlock, 0, renderStream>>>(
            inputDepTex2[0], inputDepTex2[1], FAVRSubsampleWidths[FAVRIdx]);

        blockPerGrid.x = (renderSz.x + threadPerBlock.x - 1) / threadPerBlock.x;
        reconsKernel<<<blockPerGrid, threadPerBlock, 0, renderStream>>>(
            outputSurf2[0], outputSurf2[1]);
    } break;
    case RenderTarget::FAVRSubsampleCoord: {
        dim3 threadPerBlock{16, 16};
        dim3 blockPerGrid{
            (FAVRSubsampleWidths[FAVRIdx] + threadPerBlock.x - 1) /
                threadPerBlock.x,
            (renderSz.y + threadPerBlock.y - 1) / threadPerBlock.y, 2};
        testSubsampleKernel<<<blockPerGrid, threadPerBlock, 0, renderStream>>>(
            outputSurf2[0], outputSurf2[1], FAVRSubsampleWidths[FAVRIdx]);
    } break;
    case RenderTarget::FAVRReconsCoord: {
        dim3 threadPerBlock{16, 16};
        dim3 blockPerGrid{
            (renderSz.x + threadPerBlock.x - 1) / threadPerBlock.x,
            (renderSz.y + threadPerBlock.y - 1) / threadPerBlock.y, 2};
        testReconsKernel<<<blockPerGrid, threadPerBlock, 0, renderStream>>>(
            outputSurf2[0], outputSurf2[1]);
    } break;
    default:
        break;
    }

    for (uint8_t eyeIdx = 0; eyeIdx < 2; ++eyeIdx) {
        hipDestroySurfaceObject(outputSurf2[eyeIdx]);
        hipGraphicsUnmapResources(1, &outputSurfRsrc2[eyeIdx], renderStream);

        hipDestroyTextureObject(inputDepTex2[eyeIdx]);
        hipGraphicsUnmapResources(1, &inputDepTexRsrc2[eyeIdx], renderStream);
    }
}
