#include "hip/hip_runtime.h"
#include "comp_vol_algorithm.h"
#include "comp_vol_vr_renderer.h"

#include <vs_core/VolumeSlicer/helper.hpp>

#include <thrust/async/reduce.h>
#include <thrust/device_vector.h>

#include <>

using namespace kouek;

struct BufElem {
    glm::vec3 pos;
    float scalar;

    __host__ __device__ bool operator<(const BufElem& other) const {
        return this->scalar < other.scalar;
    }
};

extern __constant__ CompVolVRRenderer::VolumeParam dc_volumeParam;

struct MaxVoxParam {
    glm::uvec3 sampleBoxSz;
    float minScalar;
};
static MaxVoxParam maxVoxParam;
static __constant__ MaxVoxParam dc_maxVoxParam;

static const dim3 threadPerBlock{16, 16};

static thrust::device_vector<BufElem> YZBuf;

static hipStream_t algorithmStream = nullptr;
static thrust::future<thrust::cuda_cub::execute_on_stream, BufElem>
    maxPosFuture;

void kouek::prepareMaxVoxPos(const glm::uvec3 &sampleBoxSzVRSp,
                             float minScalar) {
    if (algorithmStream == nullptr)
        CUDA_RUNTIME_CHECK(hipStreamCreate(&algorithmStream));

    maxVoxParam.sampleBoxSz = sampleBoxSzVRSp;
    maxVoxParam.minScalar = minScalar;
    CUDA_RUNTIME_API_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dc_maxVoxParam), &maxVoxParam, sizeof(maxVoxParam)));
    YZBuf.resize((size_t)sampleBoxSzVRSp.y * sampleBoxSzVRSp.z);
}

extern __device__ float virtualSampleLOD0(const glm::vec3 &samplePos);

__global__ void maxXYZVoxKernel(BufElem *outBuf, glm::vec3 rangeVRSp,
                                glm::vec3 minVRSp) {
    glm::uvec2 yz{blockIdx.x * blockDim.x + threadIdx.x,
                  blockIdx.y * blockDim.y + threadIdx.y};
    if (yz[0] >= dc_maxVoxParam.sampleBoxSz.y ||
        yz[1] >= dc_maxVoxParam.sampleBoxSz.z)
        return;

    BufElem bufElem;
    bufElem.scalar = 0.f;
    bufElem.pos = glm::vec3{NONE_POS_VAL};

    auto step3 = rangeVRSp;
    step3.x /= (float)dc_maxVoxParam.sampleBoxSz.x;
    step3.y /= (float)dc_maxVoxParam.sampleBoxSz.y;
    step3.z /= (float)dc_maxVoxParam.sampleBoxSz.z;

    auto pos = minVRSp;
    pos.y += step3.y * yz[0];
    pos.z += step3.z * yz[1];
    for (glm::uint stepCnt = 0; stepCnt < dc_maxVoxParam.sampleBoxSz.x;
         ++stepCnt, pos.x += step3.x) {
        auto samplePos = pos / dc_volumeParam.spaces;
        auto scalar = virtualSampleLOD0(samplePos);
        if (scalar < dc_maxVoxParam.minScalar)
            continue;
        
        if (bufElem.scalar < scalar) {
            bufElem.scalar = scalar;
            bufElem.pos = pos;
        }
    }

    outBuf[(size_t)yz[1] * dc_maxVoxParam.sampleBoxSz.z + yz[0]] = bufElem;
}

void kouek::execMaxVoxPos(const glm::vec3 &minVRSp, const glm::vec3 &maxVRSp) {
    dim3 blockPerGrid{
        (maxVoxParam.sampleBoxSz.y + threadPerBlock.x - 1) / threadPerBlock.x,
        (maxVoxParam.sampleBoxSz.z + threadPerBlock.y - 1) / threadPerBlock.y};
    maxXYZVoxKernel<<<blockPerGrid, threadPerBlock, 0, algorithmStream>>>(
        thrust::raw_pointer_cast(YZBuf.data()), maxVRSp - minVRSp, minVRSp);

    BufElem initVal{glm::vec3{NONE_POS_VAL}, 0.f};
    maxPosFuture =
        thrust::async::reduce(thrust::device.on(algorithmStream), YZBuf.begin(),
                              YZBuf.end(), initVal, thrust::maximum<BufElem>());
}

std::tuple<bool, glm::vec3> kouek::fetchMaxVoxPos() {
    if (maxPosFuture.ready())
        return std::make_tuple(true, maxPosFuture.extract().pos);
    return std::make_tuple(false, glm::zero<glm::vec3>());
}

void kouek::waitForAllVoxAlgorithms() {
    hipStreamSynchronize(algorithmStream);
    maxPosFuture = decltype(maxPosFuture)(); // reset
    hipStreamDestroy(algorithmStream);
}
